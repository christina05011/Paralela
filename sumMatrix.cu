#include <cstdio>
#include <hip/hip_runtime.h>
#include <stdlib.h>

using namespace std;

#if defined(NDEBUG)
#define CUDA_CHECK (x) (x)
#else
#define CUDA_CHECK(X) do{\
	(X);\
	hipError_t e = hipGetLastError(); \
	if(hipSuccess != e){\
		printf("cuda failure %s at %s : %d", hipGetErrorString(e), __FILE__, __LINE__);\
		exit(1);\
	}\
}while (0)
#endif

__global__ void addKernel(int* c, const int* a, const int* b)
{
	int x = threadIdx.x;
	int y = threadIdx.y;
	int i = y * (blockDim.x) + x;	//[y][x] = y * width + x
	c[i] = a[i] + b[i];
}

int main()
{
	const int WIDTH = 5;
	int a[WIDTH][WIDTH];
	int b[WIDTH][WIDTH];
	int c[WIDTH][WIDTH] = { 0 };

	//make a,b matrix
	for (int y = 0; y < WIDTH; y++)
	{
		for (int x = 0; x < WIDTH; x++)
		{
			a[y][x] = rand() % 20;
			b[y][x] = rand() % 20;
		}
	}

    //imprimiendo matriz A
    printf("VALORES DE MATRIZ A \n");
	for (int y = 0; y < WIDTH; y++)
	{
		for (int x = 0; x < WIDTH; x++)
		{
			printf("%5d", a[y][x]);
		}
		printf("\n");
	}
	
	printf("\n\n VALORES DE MATRIZ A \n");
	for (int y = 0; y < WIDTH; y++)
	{
		for (int x = 0; x < WIDTH; x++)
		{
			printf("%5d", b[y][x]);
		}
		printf("\n\n");
	}
	
	// device-side data
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;

	//allocate device memory
	CUDA_CHECK(hipMalloc((void**)&dev_a, WIDTH * WIDTH * sizeof(int)));
	CUDA_CHECK(hipMalloc((void**)&dev_b, WIDTH * WIDTH * sizeof(int)));
	CUDA_CHECK(hipMalloc((void**)&dev_c, WIDTH * WIDTH * sizeof(int)));

	//copy from host to device
	CUDA_CHECK(hipMemcpy(dev_a, a, WIDTH * WIDTH * sizeof(int), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(dev_b, b, WIDTH * WIDTH * sizeof(int), hipMemcpyHostToDevice));

	//launch a kernel on the GPU with one thread for each element
	dim3 dimBlock(WIDTH, WIDTH, 1);
	addKernel <<<1, dimBlock >>> (dev_c, dev_a, dev_b);
	CUDA_CHECK(hipPeekAtLastError());
	
	//copy from device to host
	CUDA_CHECK(hipMemcpy(c, dev_c, WIDTH * WIDTH * sizeof(int), hipMemcpyDeviceToHost));

	//free device memory
	CUDA_CHECK(hipFree(dev_c));
	CUDA_CHECK(hipFree(dev_a));
	CUDA_CHECK(hipFree(dev_b));

	//print the result
	printf("RESULTADO DE SUMA \n");
	for (int y = 0; y < WIDTH; y++)
	{
		for (int x = 0; x < WIDTH; x++)
		{
			printf("%5d", c[y][x]);
		}
		printf("\n");
	}
	return 0;
}